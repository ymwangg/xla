#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <iostream>

// #include "custom.h"
#include "tensorflow/compiler/xla/service/custom_call_target_registry.h"
#include "tensorflow/compiler/xla/xla_data.pb.h"

void xla_upsample_nearest2d(hipStream_t stream, void** buffers, const char* opaque,
                            size_t opaque_len) {
  std::cout << "xla_upsample_nearest2d" << std::endl;
  // float* output = reinterpret_cast<float*>(buffers[0]);
  // xla::ShapeProto shape;
  // shape.ParseFromArray(opaque, opaque_len);
  // int64_t len = 1;
  // for (size_t i = 0; i < shape.dimensions().size(); i++) {
  //   len *= shape.dimensions(i);
  //   std::cout << shape.dimensions(i) << std::endl;
  // }
  // std::call_once(cuda_rng, []() {
  //   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  //   hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  // });
  // hiprandSetStream(gen, stream);
  // hiprandGenerateUniform(gen, output, len);
  // std::cout << "done rng" << std::endl;
  // thrust::device_ptr<float> o = thrust::device_pointer_cast(output);
  // thrust::transform(thrust::cuda::par.on(stream), o, o + len, o,
  // bernoulli_fast(0.1));
}
XLA_REGISTER_CUSTOM_CALL_TARGET_WITH_SYM("ResizeNearest",
                                         xla_upsample_nearest2d, "CUDA");

void xla_upsample_nearest2d_backward(hipStream_t stream, void** buffers,
                                     const char* opaque, size_t opaque_len) {
  std::cout << "xla_upsample_nearest2d_backward" << std::endl;
  // float* output = reinterpret_cast<float*>(buffers[0]);
  // xla::ShapeProto shape;
  // shape.ParseFromArray(opaque, opaque_len);
  // int64_t len = 1;
  // for (size_t i = 0; i < shape.dimensions().size(); i++) {
  //   len *= shape.dimensions(i);
  //   std::cout << shape.dimensions(i) << std::endl;
  // }
  // std::call_once(cuda_rng, []() {
  //   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  //   hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  // });
  // hiprandSetStream(gen, stream);
  // hiprandGenerateUniform(gen, output, len);
  // std::cout << "done rng" << std::endl;
  // thrust::device_ptr<float> o = thrust::device_pointer_cast(output);
  // thrust::transform(thrust::cuda::par.on(stream), o, o + len, o,
  // bernoulli_fast(0.1));
}

XLA_REGISTER_CUSTOM_CALL_TARGET_WITH_SYM("ResizeNearestGrad",
                                         xla_upsample_nearest2d_backward,
                                         "CUDA");